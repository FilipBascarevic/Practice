
#include <hip/hip_runtime.h>
#include <stdio.h>

#define RADIUS 2
#define BLOCK_SIZE 10
#define BLOCK_COUNT 2

__global__ void stencil_1d(int *in, int *out) {

	// shared memory
	__shared__ int temp[BLOCK_SIZE + 2*RADIUS];
	
	// element in array
	int gindex = threadIdx.x + blockIdx.x * blockDim.x;
	// element in shared memory
	int lindex = threadIdx.x + RADIUS;
	
	// Read input elements into shared memory
	temp[lindex] = in[gindex];
	
	if (threadIdx.x < RADIUS) {
		temp[lindex-RADIUS] = in[gindex-RADIUS];
		temp[lindex+BLOCK_SIZE] = in[gindex+BLOCK_SIZE];
	}
	
	// Synchronize (ensure all the data is available)
	__syncthreads();
	
	// Apply the stencil
	int result = 0;
	for(int offset = -RADIUS; offset <= RADIUS; offset++) {
		result += temp[lindex+offset];
	}
	
	// Store the result
	out[gindex] = result;
	
}

int main (void) {

	int *input;
	int *output;
	
	int *d_input;
	int *d_output;
	
	// input array
	input = (int *)malloc((BLOCK_SIZE*BLOCK_COUNT + 2*RADIUS)*sizeof(int));
	for (int index = 0; index < BLOCK_SIZE*BLOCK_COUNT + 2*RADIUS; index++) {
		input[index] = index;
	}
	// output array
	output = (int *)malloc((BLOCK_SIZE*BLOCK_COUNT)*sizeof(int));
	
	// create array in device
	hipMalloc((void **)&d_input, (BLOCK_SIZE*BLOCK_COUNT + 2*RADIUS)*sizeof(int));
	hipMalloc((void **)&d_output, (BLOCK_SIZE*BLOCK_COUNT)*sizeof(int));
	
	// copy data from host to device
	hipMemcpy(d_input, input, (BLOCK_SIZE*BLOCK_COUNT + 2*RADIUS)*sizeof(int), hipMemcpyHostToDevice);
	
	// Run kernel
	stencil_1d<<<BLOCK_COUNT,BLOCK_SIZE>>>(d_input+RADIUS, d_output);
	
	// copy data from device to host
	hipMemcpy(output, d_output, (BLOCK_SIZE*BLOCK_COUNT)*sizeof(int), hipMemcpyDeviceToHost);
	
	hipFree(d_input);
	hipFree(d_output);
	
	// Display result
	printf("Result of 1D stencil is : \n");
	for (int index = 0; index < BLOCK_SIZE*BLOCK_COUNT; index++) {
		printf("%d ", output[index]);
	}
	printf("\n");
	
	
	
	free(input);
	free(output);
	
}
